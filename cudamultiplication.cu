#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

// Fixed kernel declaration with proper double underscores
__global__ void matrixMul(int* A, int* B, int* C, int N) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (r < N && c < N) {
        int sum = 0;
        for (int k = 0; k < N; ++k)
            sum += A[r * N + k] * B[k * N + c];
        C[r * N + c] = sum;
    }
}

// Error checking helper function
inline void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    int N;
    cout << "Enter matrix size N (NxN): ";
    cin >> N;
    
    // Validate input
    if (N <= 0) {
        cerr << "Error: Matrix size must be positive" << endl;
        return EXIT_FAILURE;
    }
    
    int size = N * N * sizeof(int);
    int *A = new int[N * N], *B = new int[N * N], *C = new int[N * N];
    
    cout << "Enter matrix A (" << N << "x" << N << "):\n";
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cout << "A[" << i << "][" << j << "]: ";
            cin >> A[i * N + j];
        }
    }
    
    cout << "Enter matrix B (" << N << "x" << N << "):\n";
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cout << "B[" << i << "][" << j << "]: ";
            cin >> B[i * N + j];
        }
    }
    
    int *dA, *dB, *dC;
    checkCudaError(hipMalloc(&dA, size), "Failed to allocate device memory for A");
    checkCudaError(hipMalloc(&dB, size), "Failed to allocate device memory for B"); 
    checkCudaError(hipMalloc(&dC, size), "Failed to allocate device memory for C");
    
    checkCudaError(hipMemcpy(dA, A, size, hipMemcpyHostToDevice), 
                  "Failed to copy A from host to device");
    checkCudaError(hipMemcpy(dB, B, size, hipMemcpyHostToDevice),
                  "Failed to copy B from host to device");
    
    dim3 threads(16, 16);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);
    
    // Added timing measurement
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "Failed to create start event");
    checkCudaError(hipEventCreate(&stop), "Failed to create stop event");
    
    checkCudaError(hipEventRecord(start), "Failed to record start event");
    
    matrixMul<<<blocks, threads>>>(dA, dB, dC, N);
    
    // Check for kernel launch errors
    checkCudaError(hipGetLastError(), "Kernel launch failed");
    checkCudaError(hipDeviceSynchronize(), "Kernel execution failed");
    
    checkCudaError(hipEventRecord(stop), "Failed to record stop event");
    checkCudaError(hipEventSynchronize(stop), "Failed to synchronize stop event");
    
    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "Failed to get elapsed time");
    
    checkCudaError(hipMemcpy(C, dC, size, hipMemcpyDeviceToHost),
                  "Failed to copy C from device to host");
    
    cout << "\nResult Matrix C:\n";
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            cout << C[i * N + j] << " ";
        cout << "\n";
    }
    
    cout << "\nMatrix multiplication completed in " << milliseconds << " ms\n";
    
    // Free resources
    hipFree(dA); 
    hipFree(dB); 
    hipFree(dC);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    delete[] A; 
    delete[] B; 
    delete[] C;
    
    return 0;
}